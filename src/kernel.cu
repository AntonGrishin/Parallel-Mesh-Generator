#include "hip/hip_runtime.h"
/*
* Name: cuda_pip.cuh
* Author  : Evgenii Vasilev
* Created : 05.04.2016
* Description: Main function in mesher program
* Version: 1.0
*/

#include "hip/hip_runtime.h"
#include ""
#include "cuda_pip.cuh"
#include "cuda_meshgen.cuh"
#include "helper_math.cuh"

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <map>

#include "grainmesh.h"
#include "iotetgen.h"
#include "mystl.h"
#include "mymesh.h"
#include "meshcut.h"
#include "meshsmooth.h"
#include <ctime>

#define VarToStr(v) #v
using grainSts = GRAIN_STATUS;

void convertMeshToArrays(grain::GrainMesh* mesh, float3* &vertices, std::vector<float> * tr, float3* &triangles);

void printHelp() {
    std::cout << "You can use following flags: " << std::endl;
    std::cout << " -nX (nY, nZ) to specify mesh size (optional) " << std::endl;
    std::cout << " -offX (offY, offZ) to specify mesh offset (optional)" << std::endl;
    std::cout << " -edgeLen to specify edge length (optional, default 0.2) " << std::endl;
    std::cout << " -generateMeshGPU to generate mesh on GPU (optional, default true) " << std::endl;
    std::cout << " -saveMeshAfterGenerate to save generated mesh (optional, default true) " << std::endl;
    std::cout << " -loadMeshBeforeMark load generated mesh for mark (optional, default false) " << std::endl;
    std::cout << " -markMeshGPU to mark mesh on GPU (optional, default true) " << std::endl;
    std::cout << " -saveMeshAfterMark to save marked mesh (optional, default true) " << std::endl;
    std::cout << " -loadMeshBeforeCut to load marked mesh before Cut (optional, default false) " << std::endl;
    std::cout << " -cutMesh to cut mesh(optional, default true) " << std::endl;
    std::cout << " -saveMeshAfterCut to save mesh generation (optional, default true) " << std::endl;
    std::cout << " -loadMeshBeforeSmooth to load cut mesh before smoth (optional, default false) " << std::endl;
    std::cout << " -smoothMesh to start mesh smooth (optional, default true) " << std::endl;
    std::cout << " -saveMeshAfterSmooth to save marked mesh afterSmooth (optional, default true) " << std::endl;
    std::cout << " -i path to input STL file (optional, default hardcode) " << std::endl;
    std::cout << " -o path to results and tmp files (optional, default hardcode) " << std::endl;
}

bool Validate(const int &lhs, const int &rhs) {
    return lhs >= rhs;
}

grainSts ValidateAndAddtoMap(char * input_args[], const int& count_arg, int &i, std::map<std::string, std::string>& args) {
    if (Validate(i + 1, count_arg)) {
        return GRAIN_ERR_WRONG_PARAMETER;
    }
    else {
        args[input_args[i]] = input_args[++i];
        return GRAIN_ERR_NONE;
    }
}

void AddSimpleFlag(char * input_args[], const int& count_arg, int &i, std::map<std::string, std::string>& args) {
    args[input_args[i]] = "1";
}

grainSts parseInputFlags(char* input_args[], const int& count_arg, std::map<std::string, std::string>& args) {
    if (count_arg == 1) {
        std::cout << "Run without parameters, set to defaults..." << std::endl;
        return GRAIN_ERR_NONE;
    }

    for (int i = 0; i < count_arg; i++) {

        if (input_args[i] == "-h") {
            printHelp();
            return GRAIN_ERR_NONE;
        }
        else if (input_args[i] == "-nX") {
            if (ValidateAndAddtoMap(input_args, count_arg, i, args) == GRAIN_ERR_WRONG_PARAMETER)
                return GRAIN_ERR_WRONG_PARAMETER;
        }
        else if (input_args[i] == "-nY") {
            if (ValidateAndAddtoMap(input_args, count_arg, i, args) == GRAIN_ERR_WRONG_PARAMETER)
                return GRAIN_ERR_WRONG_PARAMETER;
        }
        else if (input_args[i] == "-nZ") {
            if (ValidateAndAddtoMap(input_args, count_arg, i, args) == GRAIN_ERR_WRONG_PARAMETER)
                return GRAIN_ERR_WRONG_PARAMETER;
        }
        else if (input_args[i] == "-offX") {
            if (ValidateAndAddtoMap(input_args, count_arg, i, args) == GRAIN_ERR_WRONG_PARAMETER)
                return GRAIN_ERR_WRONG_PARAMETER;
        }
        else if (input_args[i] == "-offY") {
            if (ValidateAndAddtoMap(input_args, count_arg, i, args) == GRAIN_ERR_WRONG_PARAMETER)
                return GRAIN_ERR_WRONG_PARAMETER;
        }
        else if (input_args[i] == "-offZ") {
            if (ValidateAndAddtoMap(input_args, count_arg, i, args) == GRAIN_ERR_WRONG_PARAMETER)
                return GRAIN_ERR_WRONG_PARAMETER;
        }
        else if (input_args[i] == "-generateMeshGPU") {
            AddSimpleFlag(input_args, count_arg, i, args);
        }
        else if (input_args[i] == "-saveMeshAfterGenerate") {
            AddSimpleFlag(input_args, count_arg, i, args);
        }
        else if (input_args[i] == "-loadMeshBeforeMark") {
            AddSimpleFlag(input_args, count_arg, i, args);
        }
        else if (input_args[i] == "-markMeshGPU") {
            AddSimpleFlag(input_args, count_arg, i, args);
        }
        else if (input_args[i] == "-saveMeshAfterMark") {
            AddSimpleFlag(input_args, count_arg, i, args);
        }
        else if (input_args[i] == "-loadMeshBeforeCut") {
            AddSimpleFlag(input_args, count_arg, i, args);
        }
        else if (input_args[i] == "-cutMesh") {
            AddSimpleFlag(input_args, count_arg, i, args);
        }
        else if (input_args[i] == "-saveMeshAfterCut") {
            AddSimpleFlag(input_args, count_arg, i, args);
        }
        else if (input_args[i] == "-loadMeshBeforeSmooth") {
            AddSimpleFlag(input_args, count_arg, i, args);
        }
        else if (input_args[i] == "-smoothMesh") {
            AddSimpleFlag(input_args, count_arg, i, args);
        }
        else if (input_args[i] == "-saveMeshAfterSmooth") {
            AddSimpleFlag(input_args, count_arg, i, args);
        }
        else if (input_args[i] == "-i") {
            if (ValidateAndAddtoMap(input_args, count_arg, i, args) == GRAIN_ERR_WRONG_PARAMETER)
                return GRAIN_ERR_WRONG_PARAMETER;
        }
        else if (input_args[i] == "-o") {
            if (ValidateAndAddtoMap(input_args, count_arg, i, args) == GRAIN_ERR_WRONG_PARAMETER)
                return GRAIN_ERR_WRONG_PARAMETER;
        }
        else {
            return GRAIN_ERR_WRONG_PARAMETER;
        }
    }
    return GRAIN_ERR_NONE;
}

std::string getDefault(std::string param_name) {
    std::map<std::string, std::string> default_flags = {
        {"nX", "75"}, {"nY", "75"}, {"nZ", "75"},
        { "offX", "0.0" }, { "offX", "0.0" }, { "offZ", "0.0" }, {"egdeLen", "3"},
        { "generateMeshGPU"," 1" },
        { "saveMeshAfterGenerate"," 1" },
        { "loadMeshBeforeMark"," 0" },
        { "markMeshGPU"," 0" },
        { "saveMeshAfterMark"," 0" },
        { "loadMeshBeforeCut"," 0" },
        { "cutMesh"," 0" },
        { "saveMeshAfterCut"," 0" },
        { "loadMeshBeforeSmooth"," 0" },
        { "smoothMesh"," 0" },
        { "saveMeshAfterSmooth"," 0" },
        { "i"," D:\\study\\Graphics\\Parallel-Mesh-Generator\\stl\\00_heart_shell.stl" },
        { "o"," D:\\study\\Graphics\\node\\" }
    };
    return default_flags[param_name];
}

template <class T>
T convert_to_template(const std::string &str) {
    std::istringstream ss(str);
    T num;
    ss >> num;
    return num;
}

template <class T>
void SetParams(std::map<std::string, std::string> flags, T& param, std::string param_name) {
    param = convert_to_template<T>((flags["-" + param_name] == "") ? getDefault(param_name) : flags["-" + param_name]);
}

int main(int argc, char* argv[]) {
    ///
    /// Change parameters here ///
    ///
    std::cout << "Parsing parameters" << std::endl;
    std::map<std::string, std::string> flags;
    // Filename of heart shell // 
    std::string fileHeartStl;
    //fileHeartStl = "E:/Data/STL/CyberheartModel/00_heart_shell.stl";
    // Folder to store mesh results //
    std::string folderpath;
    grainSts sts;
    sts = parseInputFlags(argv, argc, flags);

    // Set mesh parameters //
    int nX, nY, nZ;
    float offX, offY, offZ;
    float edgeLen;

    bool generateMeshGPU;
    bool saveMeshAfterGenerate;
    bool loadMeshBeforeMark;
    bool markMeshGPU;
    bool saveMeshAfterMark;
    bool loadMeshBeforeCut;
    bool cutMesh;
    bool saveMeshAfterCut;
    bool loadMeshBeforeSmooth;
    bool smoothMesh;
    bool saveMeshAfterSmooth;

    if (sts == GRAIN_ERR_NONE) {
        SetParams(flags, nX, VarToStr(nX));
        SetParams(flags, nY, VarToStr(nY));
        SetParams(flags, nZ, VarToStr(nZ));
        SetParams(flags, offX, VarToStr(offX));
        SetParams(flags, offY, VarToStr(offY));
        SetParams(flags, offZ, VarToStr(offZ));
        SetParams(flags, edgeLen, VarToStr(edgeLen));
        SetParams(flags, generateMeshGPU, VarToStr(generateMeshGPU));
        SetParams(flags, saveMeshAfterGenerate, VarToStr(saveMeshAfterGenerate));
        SetParams(flags, loadMeshBeforeMark, VarToStr(loadMeshBeforeMark));
        SetParams(flags, markMeshGPU, VarToStr(markMeshGPU));
        SetParams(flags, saveMeshAfterMark, VarToStr(saveMeshAfterMark));
        SetParams(flags, loadMeshBeforeCut, VarToStr(loadMeshBeforeCut));
        SetParams(flags, cutMesh, VarToStr(cutMesh));
        SetParams(flags, saveMeshAfterCut, VarToStr(saveMeshAfterCut));
        SetParams(flags, loadMeshBeforeSmooth, VarToStr(loadMeshBeforeSmooth));
        SetParams(flags, smoothMesh, VarToStr(smoothMesh));
        SetParams(flags, saveMeshAfterSmooth, VarToStr(saveMeshAfterSmooth));
        SetParams(flags, fileHeartStl, VarToStr(i));
        SetParams(flags, folderpath, VarToStr(o));
    }
    else if (sts == GRAIN_ERR_WRONG_PARAMETER) {
        std::cout << "Wrong parameters" << std::endl;
        printHelp();
        return 0;
    }


    ///
    /// Change parameters here  ///
    ///


    int pCount = nX * nY * nZ;
    int tCount = (nX - 1)*(nY - 1)*(nZ - 1) * 6;
    MyMesh mymesh;
    MySTL stl; stl.readSTL(fileHeartStl);


    float3 *dev_points = 0;
    int4 *dev_tetra = 0;

    clock_t  timeMeshGenStart = 0, timeMeshGenEnd = 0,
        timeMeshMarkStart = 0, timeMeshMarkEnd = 0,
        timeMeshCutStart = 0, timeMeshCutEnd = 0,
        timeMeshSmoothStart = 0, timeMeshSmoothEnd = 0;

    // Generate mesh // 
    if (generateMeshGPU) {
        std::cout << "Start meshgen..." << std::endl;
        // Generate mesh with CUDA //
        timeMeshGenStart = clock();
        hipError_t cudaStatus = genMeshWithCuda(dev_points, dev_tetra,
            nX, nY, nZ, offX, offY, offZ, edgeLen);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "genMeshWithCuda failed!");
            return 1;
        }
        timeMeshGenEnd = clock();
        std::cout << "Meshgen finished successful!" << std::endl;
    }

    // Save mesh after generate //
    if (saveMeshAfterGenerate) {
        std::cout << "Start saving generated mesh..." << std::endl;
        // Copy Mesh from GPU and save in file //
        float3 *points = new float3[pCount];
        int4 *tetra = new int4[tCount];
        copyMeshFromGPU(points, dev_points, pCount,
            tetra, dev_tetra, tCount);

        // Save mesh //
        MyMesh mymesh;
        mymesh.mPoints = points;
        mymesh.mPointsCount = pCount;
        short* pLabels = new short[pCount];
        for (int i = 0; i < pCount; i++)
            pLabels[i] = 0;
        mymesh.mPointLabels = pLabels;

        mymesh.mTetra = tetra;
        mymesh.mTetraCount = tCount;
        short* tLabels = new short[tCount];
        for (int i = 0; i < tCount; i++)
            tLabels[i] = 0;
        mymesh.mTetraLabels = tLabels;

        grain::saveNodeFile(folderpath + "meshGenerated.node", &mymesh);
        grain::saveEleFile(folderpath + "meshGenerated.ele", &mymesh);
        std::cout << "Mesh successful saved to " <<
            folderpath << "meshGenerated.node" << std::endl;
    }

    if (loadMeshBeforeMark) {
        grain::readNodeFile(folderpath + "torscoloredremoved.node", &mymesh);
        //grain::readEleFile(folderpath + "torscoloredremoved.ele", &mymesh);

        pCount = mymesh.mPointsCount;
        //tCount = mymesh.mTetraCount;

        hipError_t cudaStatus;
        // Choose which GPU to run on, change this on a multi-GPU system.
        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed? \n");
        }
        // Allocate GPU buffers for points
        cudaStatus = hipMalloc((void**)&dev_points, pCount * sizeof(float3));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed! \n");
        }
        float3* p = mymesh.mPoints;
        // Copy input vectors from host memory to GPU buffers.
        cudaStatus = hipMemcpy(dev_points, p,
            pCount * sizeof(float3), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy 1  failed! \n");
        }

        // Copy input vectors from host memory to GPU buffers.
        //cudaStatus = hipMemcpy(dev_tetra, mymesh.mTetra,
        //	tCount * sizeof(int4), hipMemcpyHostToDevice);
        //if (cudaStatus != hipSuccess) {
        //	fprintf(stderr, "hipMemcpy 1  failed! \n");
        //}

    }

    // Mark labels with CUDA // 
    if (markMeshGPU) {
        float timeWithCopy, timeWithoutCopy;
        float3* mystl = new float3[stl.trigs.size() / 3];
        for (uint i = 0; i < stl.trigs.size() / 3; i++) {
            mystl[i].x = stl.trigs[3 * i + 0];
            mystl[i].y = stl.trigs[3 * i + 1];
            mystl[i].z = stl.trigs[3 * i + 2];
        }
        bool * result = new bool[pCount];
        timeMeshMarkStart = clock();
        // Mark mesh with CUDA //
        hipError_t cudaStatus = calcIntersectionCuda2(result,
            dev_points, pCount,
            mystl, stl.trigs.size() / 9,
            timeWithCopy, timeWithoutCopy);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "calcIntersectionCuda failed! \n");
            return 1;
        }
        timeMeshMarkEnd = clock();

        // Generate mesh with labels //
        short* resvec = new short[pCount];
        for (int i = 0; i < pCount; i++) {
            int val = 999;
            if (result[i] == true)
                val = 0;
            resvec[i] = val;
        }

        /// Copy Mesh from GPU and save in file //

        float3 *points = new float3[pCount];
        int4 *tetra = new int4[tCount];
        copyMeshFromGPU(points, dev_points, pCount,
            tetra, dev_tetra, tCount);

        mymesh.mPoints = points;
        mymesh.mPointsCount = pCount;
        mymesh.mPointLabels = resvec;

        mymesh.mTetra = tetra;
        mymesh.mTetraCount = tCount;
        short* tLabels = new short[tCount];
        for (int i = 0; i < tCount; i++)
            tLabels[i] = 0;
        mymesh.mTetraLabels = tLabels;
    }

    // Save mesh after mark // 
    if (saveMeshAfterMark) {
        grain::saveNodeFile(folderpath + "meshMarked.node", &mymesh);
        grain::saveEleFile(folderpath + "meshMarked.ele", &mymesh);
    }

    // Load mesh before cut // 
    if (loadMeshBeforeCut) {
        grain::readNodeFile(folderpath + "meshMarked.node", &mymesh);
        grain::readEleFile(folderpath + "meshMarked.ele", &mymesh);
    }

    // Mesh cutting //
    if (cutMesh) {
        timeMeshCutStart = clock();
        MeshCut cut;
        cut.cutMeshMarkedVertices(&mymesh);
        timeMeshCutEnd = clock();
    }

    // Save mesh before cut //
    if (saveMeshAfterCut) {
        grain::saveNodeFile(folderpath + "afterCut.node", &mymesh);
        grain::saveEleFile(folderpath + "afterCut.ele", &mymesh);
    }

    // Load mesh before smooth //
    if (loadMeshBeforeSmooth) {
        grain::readNodeFile(folderpath + "afterCut.node", &mymesh);
        grain::readEleFile(folderpath + "afterCut.ele", &mymesh);
    }

    // Smoothing //
    if (smoothMesh) {
        timeMeshSmoothStart = clock();
        MeshSmooth smooth;
        smooth.edgelen = edgeLen;
        smooth.smoothMesh(&mymesh, &stl);
        timeMeshSmoothEnd = clock();
    }

    // Save smooth after smooth //
    if (saveMeshAfterSmooth) {
        grain::saveNodeFile(folderpath + "afterSmooth.node", &mymesh);
        grain::saveEleFile(folderpath + "afterSmooth.ele", &mymesh);
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    /*cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed! \n");
        return 1;
    }*/

    std::ofstream fout(folderpath + "result.txt");
    fout << " Time mesh generate " << static_cast<float>(timeMeshGenEnd - timeMeshGenStart) / CLK_TCK << "\n"
        << " Time mesh mark " << static_cast<float>(timeMeshMarkEnd - timeMeshMarkStart) / CLK_TCK << "\n"
        << " Time mesh cut " << static_cast<float>(timeMeshCutEnd - timeMeshCutStart) / CLK_TCK << "\n"
        << " Time mesh smooth " << static_cast<float>(timeMeshSmoothEnd - timeMeshSmoothStart) / CLK_TCK << "\n";
    fout.close();

    std::cout << " Time mesh generate " << static_cast<float>(timeMeshGenEnd - timeMeshGenStart) / CLK_TCK << "\n"
        << " Time mesh mark " << static_cast<float>(timeMeshMarkEnd - timeMeshMarkStart) / CLK_TCK << "\n"
        << " Time mesh cut " << static_cast<float>(timeMeshCutEnd - timeMeshCutStart) / CLK_TCK << "\n"
        << " Time mesh smooth " << static_cast<float>(timeMeshSmoothEnd - timeMeshSmoothStart) / CLK_TCK << "\n";

    return 0;
}

void convertMeshToArrays(grain::GrainMesh* mesh, float3* &vertices, std::vector<float> * tr, float3* &triangles) {
    std::vector<vec3d>* vert = mesh->getVertices();
    vertices = new float3[mesh->getVerticesCount()];
    for (int i = 0; i < mesh->getVerticesCount(); i++) {
        vertices[i].x = vert->at(i).x;
        vertices[i].y = vert->at(i).y;
        vertices[i].z = vert->at(i).z;
    }
    triangles = new float3[tr->size() / 3];
    for (size_t i = 0; i < tr->size() / 3; i++) {
        triangles[i].x = tr->at(i * 3 + 0);
        triangles[i].y = tr->at(i * 3 + 1);
        triangles[i].z = tr->at(i * 3 + 2);
    }
}

